#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void cudaProcess(unsigned int *g_odata, int imgw) {
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;

    // if (tx == 0 and ty == 0) {
    //     printf("bw: %d, bh: %d\n", bw, bh);
    //     printf("bIdx.x: %d, bIdx.y: %d\n", blockIdx.x, blockIdx.y);
    // }

    // int block_lin_idx = blockIdx.y * gridDim.x + blockIdx.x;
    // int num_blocks = gridDim.x * gridDim.y;
    // float block_color = (float)block_lin_idx / num_blocks * 255;
    // g_odata[y * imgw + x] = rgbToInt(block_color, block_color, block_color);

    // uchar4 c4 = make_uchar4((x & 0x20) ? 255 : 0, 0, (y & 0x20) ? 255 : 0, 0);
    // g_odata[y * imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
    // uchar4 c4 = make_uchar4(blockIdx.x * 8, blockIdx.y * 8, 0, 0);

    // g_odata[y * imgw + x] = rgbToInt(c4.x, c4.y, c4.z);

    // g_odata[y * imgw + x] = rgbToInt((float)x / imgw * 255, (float)y / imgw * 255, 0);


    int iter = 0;
    int maxIter = 100;
    float maxAmp = 1.0;
    float zr = 0.0;
    float zi = 0.0;
    float zr_temp;
    float a = (float)x / imgw;
    float b = (float)y / imgw;
    while (iter < maxIter and zr*zr + zi*zi < maxAmp) {
        zr_temp = zr*zr - zi*zi + a;
        zi = 2*zr*zi + b;
        zr = zr_temp;
        iter++;
    }
    float convergence = (float)iter / maxIter;
    float color = (1.0f - convergence) * 255;
    // printf("a: %.2f, b: %.2f, conv: %.2f\n", a, b, convergence);
    g_odata[y * imgw + x] = rgbToInt(color, color, color);
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                     unsigned int *g_odata, int imgw) {
    cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw);
}
