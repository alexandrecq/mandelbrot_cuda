#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void cudaProcess(unsigned int *g_odata, int imgw) {
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;

    int imgh = imgw; //TODO passed as input
    if (x >= imgw) or (y >= imgh) {
        return
    }

    int2 screen_center = {imgw / 2, imgh / 2};
    float2 image_center = {-1.0, 0.0};
    float zoom_factor = 0.5;
    float a = (float)(x - screen_center.x) / imgw / zoom_factor + image_center.x;
    float b = (float)(y - screen_center.y) / imgh / zoom_factor + image_center.y;

    int iter = 0;
    int maxIter = 100;
    float maxAmp = 4.0;
    float zr = 0.0;
    float zi = 0.0;
    float zr_temp;
    while (iter < maxIter and zr*zr + zi*zi < maxAmp) {
        zr_temp = zr*zr - zi*zi + a;
        zi = 2*zr*zi + b;
        zr = zr_temp;
        iter++;
    }
    float convergence = (float)iter / maxIter;
    float color = (1.0f - convergence) * 255;
    g_odata[y * imgw + x] = rgbToInt(color, color, color);
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                     unsigned int *g_odata, int imgw) {
    cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw);
}

    // if (tx == 0 and ty == 0) {
    //     printf("bw: %d, bh: %d\n", bw, bh);
    //     printf("bIdx.x: %d, bIdx.y: %d\n", blockIdx.x, blockIdx.y);
    // }

    // int block_lin_idx = blockIdx.y * gridDim.x + blockIdx.x;
    // int num_blocks = gridDim.x * gridDim.y;
    // float block_color = (float)block_lin_idx / num_blocks * 255;
    // g_odata[y * imgw + x] = rgbToInt(block_color, block_color, block_color);
    // g_odata[y * imgw + x] = rgbToInt((float)x / imgw * 255, (float)y / imgw * 255, 0);

